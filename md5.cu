#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "sm_35_atomic_functions.h"
#include "md5.cuh"
#include "hip/hip_runtime_api.h"
#include <stdio.h>

#define S11 7
#define S12 12
#define S13 17
#define S14 22
#define S21 5
#define S22 9
#define S23 14
#define S24 20
#define S31 4
#define S32 11
#define S33 16
#define S34 23
#define S41 6
#define S42 10
#define S43 15
#define S44 21

#define HASH_LENGTH 16

 // the result
char *testPointer;

//declare constant memory
__constant__ unsigned char padding[64]; 
__constant__ unsigned char constant_hash[HASH_LENGTH]; //skrot, dla kt�rego b�dziemy szuka� kolizji

__device__ void checkMD5equality(unsigned char* oryginal, unsigned char * pretender , bool * equality){
	equality[0]=true;
	for(int q=0; q<16; q++){
		if(oryginal[q]!=pretender[q]){ equality[0]=false; break;}
	}
}

__device__ int checkEquality(uint1_md5 *generated_hash){
	//funkcja sprawdza r�wno�� wygenerowanego skr�tu z docelowym, na ten czas iterujemy ca�� p�tl� niesprawdzaj�c warunku equality==0 co mo�e okaza� si� gorszym rozwiazaniem
	int equality = 1;
	for(int i=0; i<HASH_LENGTH; i++){
		equality *= (generated_hash[i] == constant_hash[i]);
	}
	return equality;
}


__global__ void generateOryginalHash(uint1_md5 *mess, uint1_md5* oryginalHash){
				
				bool equal[1];
				unsigned char threadMess[128];
				unsigned char threadHash[16];
				charMemcpy(threadMess, mess, 128);
				generateMD5(threadMess, threadHash, 128);
				
				charMemcpy( oryginalHash,threadHash, 16);
			//	printf(" Obliczono wartosc skrotu oryginalnej wiadomosci ");
			
}
__global__ void generateCollision(uint1_md5 *mess, uint1_md5* oryginalHash, uint1_md5* hashCode, int* end){
				int k = threadIdx.x;
				int j = blockIdx.x;
				bool equal[1];
				unsigned char threadMess[128];
				unsigned char threadHash[16];
				
		for (int i=0; i<256 && end[0]==0; i++){
				charMemcpy(threadMess, mess, 128);

				//liczenie wektora
				threadMess[19]= mess[19]+k;
				threadMess[45]=mess[45]-j;
				threadMess[59]=mess[59]+i;

				threadMess[83]=mess[83]+k;
				threadMess[109]=mess[109]+j;
				threadMess[123]=mess[123]-i;
	
				generateMD5(threadMess, threadHash, 128);
				checkMD5equality(oryginalHash, threadHash, equal); 
		
			if(equal[0]& k!=0 && j!=0 && i!=0){
				charMemcpy( mess,threadMess, 128);
				charMemcpy(hashCode,threadHash, 16);
				//zostanie zast�pione operacj� atomow�
				//chwilowo problem z obs�ug� funkcji atomicAdd
				//zmiana warto�ci konczy wykonywanie sie wszystkich petli
					end[0]=2;				
			}
		}
		__syncthreads();
}

__device__ void printResultDevice(unsigned char* result){
	char buf[33];
//	for (int i=0; i<16; i++)
//		printf( "%02x", result[i]);
	buf[32]=0;
	//printf("%s\n", buf);
}
__device__ void charMemcpy(unsigned char *buffer, unsigned char *data, int length){

	int i;
	#pragma unroll
	for(i=0; i<length; i++){
		buffer[i] = data[i];
	}
}

/*void runMD5(unsigned char *hostPadding, unsigned char* oryginalHash, unsigned char* result){
	uint1_md5 *hashCode;
	
	unsigned char *dev_message;
	
	int *end;
	end[0]=0;
	checkCudaErrors(hipSetDevice(0));
	//kopiowanie do pamieci stalej urzadzenia
	//JR nie u�ywam narazie wiec coment
	//checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(padding), hostPadding, sizeof(char)*128, 0, hipMemcpyHostToDevice));

	//checkCudaErrors(hipMalloc((void**)&testPointer, 16*sizeof(char)));
	checkCudaErrors(hipMalloc((void**)&hashCode, 16*sizeof(char)));
	checkCudaErrors(hipMalloc((void**)&dev_oryginalHash, 16*sizeof(char)));
	checkCudaErrors(hipMalloc((void**)&dev_message, 128*sizeof(char)));
	
	checkCudaErrors(hipMalloc((void**)&dev_end, 1*sizeof(int)));
	checkCudaErrors(hipMemcpy(dev_message, hostPadding, 128 * sizeof(char), hipMemcpyHostToDevice));

	
	
	generateCollision<<<100, 100>>>(dev_message,dev_oryginalHash, hashCode, dev_end);
	checkCudaErrors(hipGetLastError());
	checkCudaErrors(hipDeviceSynchronize());

	checkCudaErrors(hipMemcpy(result, hashCode, 16*sizeof(unsigned char), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(oryginalHash, dev_oryginalHash, 16*sizeof(unsigned char), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(hostPadding, dev_message , 128 * sizeof(char), hipMemcpyDeviceToHost));
	
	checkCudaErrors(hipFree(dev_end));
	checkCudaErrors(hipFree(hashCode));
	checkCudaErrors(hipFree(dev_oryginalHash));
	checkCudaErrors(hipFree(dev_message));

	//resetowanie urz�dzenia
	checkCudaErrors(hipDeviceReset());
    return;
}*/








//declare constant memory





unsigned char hostPadding[64] = {
    0x80, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0,
    0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0, 0
  };

uint1_md5 *dev_oryginalHash;
void runMD5(unsigned char *host_message ,unsigned char* oryginalHash,  unsigned char* result,  int length){
	uint1_md5 *foundCollision; // the result
	unsigned char * message;
	int * dev_end;
	checkCudaErrors(hipSetDevice(0));
	checkCudaErrors(hipMemcpyToSymbol(HIP_SYMBOL(padding), &hostPadding, sizeof(char)*64, 0, hipMemcpyHostToDevice));
	int end[1];
	end[0]=0;

	checkCudaErrors(hipMalloc((void**)&foundCollision, 16*sizeof(char)));
	checkCudaErrors(hipMalloc((void**)&dev_oryginalHash, 16*sizeof(char)));
	checkCudaErrors(hipMalloc((void**)&message, length*sizeof(char)));
	checkCudaErrors(hipMalloc((void**)&dev_end, 1*sizeof(int)));

	checkCudaErrors(hipMemcpy(message, host_message, length*sizeof(char), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(dev_end, end, 1 * sizeof(int), hipMemcpyHostToDevice));
	generateOryginalHash<<<1, 1>>>(message,dev_oryginalHash);
	generateCollision<<<1, 1>>>(message,dev_oryginalHash, foundCollision, dev_end);
	checkCudaErrors(hipGetLastError());

	checkCudaErrors(hipMemcpy(oryginalHash, dev_oryginalHash, 16*sizeof(char), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy(result, foundCollision, 16*sizeof(char), hipMemcpyDeviceToHost));
	checkCudaErrors(hipMemcpy( host_message,message, length*sizeof(char),  hipMemcpyDeviceToHost));
	
	checkCudaErrors(hipFree(foundCollision));
	checkCudaErrors(hipFree(message));

	//resetowanie urz�dzenia
	checkCudaErrors(hipDeviceReset());
    return;
}



 
// F, G, H and I are basic MD5 functions.
__device__ uint4_md5 F(uint4_md5 x, uint4_md5 y, uint4_md5 z) {
  return x&y | ~x&z;
}
 
__device__ uint4_md5 G(uint4_md5 x, uint4_md5 y, uint4_md5 z) {
  return x&z | y&~z;
}
 
__device__ uint4_md5 H(uint4_md5 x, uint4_md5 y, uint4_md5 z) {
  return x^y^z;
}
 
__device__ uint4_md5 I(uint4_md5 x, uint4_md5 y, uint4_md5 z) {
  return y ^ (x | ~z);
}
 
// rotate_left rotates x left n bits.
__device__ uint4_md5 rotate_left(uint4_md5 x, int n) {
  return (x << n) | (x >> (32-n));
}

// FF, GG, HH, and II transformations for rounds 1, 2, 3, and 4.
// Rotation is separate from addition to prevent recomputation.
__device__ void FF(uint4_md5 &a, uint4_md5 b, uint4_md5 c, uint4_md5 d, uint4_md5 x, uint4_md5 s, uint4_md5 ac) {
  a = rotate_left(a+ F(b,c,d) + x + ac, s) + b;
}
 
__device__ void GG(uint4_md5 &a, uint4_md5 b, uint4_md5 c, uint4_md5 d, uint4_md5 x, uint4_md5 s, uint4_md5 ac) {
  a = rotate_left(a + G(b,c,d) + x + ac, s) + b;
}
 
__device__ void HH(uint4_md5 &a, uint4_md5 b, uint4_md5 c, uint4_md5 d, uint4_md5 x, uint4_md5 s, uint4_md5 ac) {
  a = rotate_left(a + H(b,c,d) + x + ac, s) + b;
}
 
__device__ void II(uint4_md5 &a, uint4_md5 b, uint4_md5 c, uint4_md5 d, uint4_md5 x, uint4_md5 s, uint4_md5 ac) {
  a = rotate_left(a + I(b,c,d) + x + ac, s) + b;
}


__device__ void transform(const uint1_md5 block[64], uint4_md5 state[4]){
	uint4_md5 a = state[0], b = state[1], c = state[2], d = state[3], x[16];

  //decode (x, block, 64);
	for (unsigned int i = 0, j = 0; j < 64; i++, j += 4)
    x[i] = ((uint4_md5)block[j]) | (((uint4_md5)block[j+1]) << 8) |
      (((uint4_md5)block[j+2]) << 16) | (((uint4_md5)block[j+3]) << 24);
  //------------koniec decode ------------------------------
 
  /* Round 1 */
  FF (a, b, c, d, x[ 0], S11, 0xd76aa478); /* 1 */
  FF (d, a, b, c, x[ 1], S12, 0xe8c7b756); /* 2 */
  FF (c, d, a, b, x[ 2], S13, 0x242070db); /* 3 */
  FF (b, c, d, a, x[ 3], S14, 0xc1bdceee); /* 4 */
  FF (a, b, c, d, x[ 4], S11, 0xf57c0faf); /* 5 */
  FF (d, a, b, c, x[ 5], S12, 0x4787c62a); /* 6 */
  FF (c, d, a, b, x[ 6], S13, 0xa8304613); /* 7 */
  FF (b, c, d, a, x[ 7], S14, 0xfd469501); /* 8 */
  FF (a, b, c, d, x[ 8], S11, 0x698098d8); /* 9 */
  FF (d, a, b, c, x[ 9], S12, 0x8b44f7af); /* 10 */
  FF (c, d, a, b, x[10], S13, 0xffff5bb1); /* 11 */
  FF (b, c, d, a, x[11], S14, 0x895cd7be); /* 12 */
  FF (a, b, c, d, x[12], S11, 0x6b901122); /* 13 */
  FF (d, a, b, c, x[13], S12, 0xfd987193); /* 14 */
  FF (c, d, a, b, x[14], S13, 0xa679438e); /* 15 */
  FF (b, c, d, a, x[15], S14, 0x49b40821); /* 16 */
 
  /* Round 2 */
  GG (a, b, c, d, x[ 1], S21, 0xf61e2562); /* 17 */
  GG (d, a, b, c, x[ 6], S22, 0xc040b340); /* 18 */
  GG (c, d, a, b, x[11], S23, 0x265e5a51); /* 19 */
  GG (b, c, d, a, x[ 0], S24, 0xe9b6c7aa); /* 20 */
  GG (a, b, c, d, x[ 5], S21, 0xd62f105d); /* 21 */
  GG (d, a, b, c, x[10], S22,  0x2441453); /* 22 */
  GG (c, d, a, b, x[15], S23, 0xd8a1e681); /* 23 */
  GG (b, c, d, a, x[ 4], S24, 0xe7d3fbc8); /* 24 */
  GG (a, b, c, d, x[ 9], S21, 0x21e1cde6); /* 25 */
  GG (d, a, b, c, x[14], S22, 0xc33707d6); /* 26 */
  GG (c, d, a, b, x[ 3], S23, 0xf4d50d87); /* 27 */
  GG (b, c, d, a, x[ 8], S24, 0x455a14ed); /* 28 */
  GG (a, b, c, d, x[13], S21, 0xa9e3e905); /* 29 */
  GG (d, a, b, c, x[ 2], S22, 0xfcefa3f8); /* 30 */
  GG (c, d, a, b, x[ 7], S23, 0x676f02d9); /* 31 */
  GG (b, c, d, a, x[12], S24, 0x8d2a4c8a); /* 32 */
 
  /* Round 3 */
  HH (a, b, c, d, x[ 5], S31, 0xfffa3942); /* 33 */
  HH (d, a, b, c, x[ 8], S32, 0x8771f681); /* 34 */
  HH (c, d, a, b, x[11], S33, 0x6d9d6122); /* 35 */
  HH (b, c, d, a, x[14], S34, 0xfde5380c); /* 36 */
  HH (a, b, c, d, x[ 1], S31, 0xa4beea44); /* 37 */
  HH (d, a, b, c, x[ 4], S32, 0x4bdecfa9); /* 38 */
  HH (c, d, a, b, x[ 7], S33, 0xf6bb4b60); /* 39 */
  HH (b, c, d, a, x[10], S34, 0xbebfbc70); /* 40 */
  HH (a, b, c, d, x[13], S31, 0x289b7ec6); /* 41 */
  HH (d, a, b, c, x[ 0], S32, 0xeaa127fa); /* 42 */
  HH (c, d, a, b, x[ 3], S33, 0xd4ef3085); /* 43 */
  HH (b, c, d, a, x[ 6], S34,  0x4881d05); /* 44 */
  HH (a, b, c, d, x[ 9], S31, 0xd9d4d039); /* 45 */
  HH (d, a, b, c, x[12], S32, 0xe6db99e5); /* 46 */
  HH (c, d, a, b, x[15], S33, 0x1fa27cf8); /* 47 */
  HH (b, c, d, a, x[ 2], S34, 0xc4ac5665); /* 48 */
 
  /* Round 4 */
  II (a, b, c, d, x[ 0], S41, 0xf4292244); /* 49 */
  II (d, a, b, c, x[ 7], S42, 0x432aff97); /* 50 */
  II (c, d, a, b, x[14], S43, 0xab9423a7); /* 51 */
  II (b, c, d, a, x[ 5], S44, 0xfc93a039); /* 52 */
  II (a, b, c, d, x[12], S41, 0x655b59c3); /* 53 */
  II (d, a, b, c, x[ 3], S42, 0x8f0ccc92); /* 54 */
  II (c, d, a, b, x[10], S43, 0xffeff47d); /* 55 */
  II (b, c, d, a, x[ 1], S44, 0x85845dd1); /* 56 */
  II (a, b, c, d, x[ 8], S41, 0x6fa87e4f); /* 57 */
  II (d, a, b, c, x[15], S42, 0xfe2ce6e0); /* 58 */
  II (c, d, a, b, x[ 6], S43, 0xa3014314); /* 59 */
  II (b, c, d, a, x[13], S44, 0x4e0811a1); /* 60 */
  II (a, b, c, d, x[ 4], S41, 0xf7537e82); /* 61 */
  II (d, a, b, c, x[11], S42, 0xbd3af235); /* 62 */
  II (c, d, a, b, x[ 2], S43, 0x2ad7d2bb); /* 63 */
  II (b, c, d, a, x[ 9], S44, 0xeb86d391); /* 64 */
 
  state[0] += a;
  state[1] += b;
  state[2] += c;
  state[3] += d;
}

__device__ void generateMD5( unsigned char * message, uint1_md5 digest[HASH_LENGTH], int length){

	uint4_md5 count[2];   // 64bit counter for number of bits (lo, hi)
	count[0] = 0;
	count[1] = 0;

	uint4_md5 state[4];		//initial values
	state[0] = 0x67452301;
	state[1] = 0xefcdab89;
	state[2] = 0x98badcfe;
	state[3] = 0x10325476;

	// update ----------------------------------------------------
	uint4_md5 index = 0;

	// Update number of bits
	if ((count[0] += (length << 3)) < (length << 3))
		count[1]++;
	count[1] += (length >> 29);

	uint4_md5 firstpart = 64;
	uint4_md5 i=0;
 
	for (i = 0; i + 64 <= length; i += 64)
      transform(&message[i], state);

	i = i * (length >= firstpart);
  
	// buffer remaining input
	uint1_md5 buffer[64]; //bytes that didn't fit in the last chunk
	charMemcpy(&buffer[index], &message[i], (length-i));

	// finalized --------------------------------------------------------------
	unsigned char bits[8];

	//encode(bits, count, 8);
	for (uint4_md5 i = 0, j = 0; j < 8; i++, j += 4) {
		bits[j] = count[i] & 0xff;
		bits[j+1] = (count[i] >> 8) & 0xff;
		bits[j+2] = (count[i] >> 16) & 0xff;
		bits[j+3] = (count[i] >> 24) & 0xff;
	}

	index = count[0] / 8 % 64;
	uint4_md5 padLen = (index < 56) ? (56 - index) : (120 - index);
  
	// compute number of bytes mod 64
	index = count[0] / 8 % 64; 

	if ((count[0] += (padLen << 3)) < (padLen << 3))
		count[1]++;
	count[1] += (padLen >> 29);

	charMemcpy(&buffer[index], &padding[0], padLen);

	//dla bits
	index = count[0] / 8 % 64; 
	firstpart = 64 - index;

	charMemcpy(&buffer[index], bits, firstpart);
	transform(buffer, state);

	// encode(digest, state, 16);
	for (uint4_md5 i = 0, j = 0; j < 16; i++, j += 4) {
		digest[j] = state[i] & 0xff;
		digest[j+1] = (state[i] >> 8) & 0xff;
		digest[j+2] = (state[i] >> 16) & 0xff;
		digest[j+3] = (state[i] >> 24) & 0xff;
	}
  //charMemcpy(foundCollision, digest, 16);
}